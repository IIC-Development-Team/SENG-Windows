﻿#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""

namespace EXT_CUDA
{
	bool IsCudaCapable()
	{
		return hipSetDevice(0) == hipSuccess;
	}
}