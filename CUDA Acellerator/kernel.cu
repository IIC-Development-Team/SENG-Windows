#include "hip/hip_runtime.h"
﻿#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""

namespace EXT_CUDA
{
	__global__ void ProcessMesh(aiMesh** pMesh, std::vector<GEO_CUDA::TempMeshData>& outData)
	{
		int t = threadIdx.x;

		for (int i = 0; i < pMesh[t]->mNumVertices; i++)
		{
			glm::vec3 v;
			glm::vec2 u;

			v.x = pMesh[t]->mVertices[i].x;
			v.y = pMesh[t]->mVertices[i].y;
			v.z = pMesh[t]->mVertices[i].z;

			outData[t].verticies.push_back(v);

			if (pMesh[t]->mTextureCoords[0]) {
				u.x = (float)pMesh[t]->mTextureCoords[0][i].x;
				u.y = (float)pMesh[t]->mTextureCoords[0][i].y;

				outData[t].uv.push_back(u);
			}
		}

		for (int i = 0; i < pMesh[t]->mNumFaces; i++)
		{
			aiFace face = pMesh[t]->mFaces[i];
			for (int j = 0; j < face.mNumIndices; j++)
			{
				outData[t].indices.push_back(face.mIndices[j]);
			}
		}
	}

	bool IsCudaCapable()
	{
		return hipSetDevice(0) == hipSuccess;
	}

	bool CUDA_EXP LoadMultipleMeshes(aiMesh** aMeshes, size_t size, std::vector<GEO_CUDA::TempMeshData>& outData)
	{
		if (!IsCudaCapable() || size <= 0 || aMeshes == nullptr)
			return false;


		if (!outData.empty())
		{
			printf("WARNING! Non empty vector was passed, all data will be errased");
			outData.clear();
		}

		for (int i = 0; i < size; i++)
		{
			GEO_CUDA::TempMeshData emptyData;
			outData.push_back(emptyData);
		}
	}
}